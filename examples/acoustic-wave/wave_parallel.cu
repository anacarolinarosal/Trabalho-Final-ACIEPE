#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <pthread.h>
#include <hip/hip_runtime.h>

#define DT 0.0070710676f // delta t
#define DX 15.0f // delta x
#define DY 15.0f // delta y
#define V 1500.0f // wave velocity v = 1500 m/s

int iterations;

int rows;
int cols;

float *swap;

float dxSquared;
float dySquared;
float dtSquared;

/*
 * save the matrix on a file.txt
 */
void save_grid(int rows, int cols, float *matrix){

    system("mkdir -p wavefield");

    char file_name[64];
    sprintf(file_name, "wavefield/wavefield_parallel15000.txt");

    // save the result
    FILE *file;
    file = fopen(file_name, "w");

    for(int i = 0; i < rows; i++) {

        int offset = i * cols;

        for(int j = 0; j < cols; j++) {
            fprintf(file, "%f ", matrix[offset + j]);
        }
        fprintf(file, "\n");
    }

    fclose(file);
    
    system("python3 plot_parallel.py");
}


__global__ void *compute_wave(float *prev_base, float *next_base, float *vel_base){

    //thread id
    int id = blockIdx.x;

    // calculate the chunk size
    int chunk = rows / num_threads;

    // calculate begin and end step of the thread
    int begin = id * chunk;
    int end = begin + chunk-1;

    // the last thread must have to end before the border
    if (id == num_threads-1)
        end = cols - 2;

    // the first thread must begin at after the beginning of the border
    if (id == 0)
        begin = begin + 1;

    // wavefield modeling
    for(int n = 0; n < iterations; n++) {
        for(int i = 1; i < rows-1; i++) {
            for(int j = begin ; j <= end; j++) {
                // index of the current point in the grid
                int current = i * cols + j;
                
                //neighbors in the horizontal direction
                float value = (prev_base[current + 1] - 2.0 * prev_base[current] + prev_base[current - 1]) / dxSquared;
                
                //neighbors in the vertical direction
                value += (prev_base[current + cols] - 2.0 * prev_base[current] + prev_base[current - cols]) / dySquared;
                
                value *= dtSquared * vel_base[current];
                
                next_base[current] = 2.0 * prev_base[current] - next_base[current] + value;
            }
        }
    }
}


int main(int argc, char* argv[]) {

    if(argc != 4){
        printf("Usage: ./stencil N1 N2 TIME\n");
        printf("N1 N2: grid sizes for the stencil\n");
        printf("TIME: propagation time in ms\n");
        exit(-1);
    }

    // number of rows of the grid
    rows = atoi(argv[1]);

    // number of columns of the grid
    cols = atoi(argv[2]);

    // number of timesteps
    int time = atoi(argv[3]);
    
    // calc the number of iterations (timesteps)
    iterations = (int)((time/1000.0) / DT);

    // Cuda error
    hipError_t syncErr, asyncErr;

    // vetores
    float *dev_prev_base;
    float *dev_next_base;
    float *dev_vel_base;

    // alocação de memória na GPU
    hipMalloc(&dev_prev_base, rows * cols * sizeof(float));
    hipMalloc(&dev_next_base, rows * cols * sizeof(float));
    hipMalloc(&dev_vel_base, rows * cols * sizeof(float));

    // alocação de memória na CPU
    float *prev_base = malloc(rows * cols * sizeof(float));
    float *next_base = malloc(rows * cols * sizeof(float));
    float *vel_base = malloc(rows * cols * sizeof(float));

    printf("Grid Sizes: %d x %d\n", rows, cols);
    printf("Iterations: %d\n", iterations);

    // ************* BEGIN INITIALIZATION *************

    printf("Initializing ... \n");

    // define source wavelet
    float wavelet[12] = {0.016387336, -0.041464937, -0.067372555, 0.386110067,
                         0.812723635, 0.416998396,  0.076488599,  -0.059434419,
                         0.023680172, 0.005611435,  0.001823209,  -0.000720549};

    // initialize matrix
    for(int i = 0; i < rows; i++){

        int offset = i * cols;

        for(int j = 0; j < cols; j++){
            prev_base[offset + j] = 0.0f;
            next_base[offset + j] = 0.0f;
            vel_base[offset + j] = V * V;
        }
    }

    // add a source to initial wavefield as an initial condition
    for(int s = 11; s >= 0; s--){
        for(int i = rows / 2 - s; i < rows / 2 + s; i++){

            int offset = i * cols;

            for(int j = cols / 2 - s; j < cols / 2 + s; j++)
                prev_base[offset + j] = wavelet[s];
        }
    }

    // ************** END INITIALIZATION **************

    printf("Computing wavefield ... \n");

    dxSquared = DX * DX;
    dySquared = DY * DY;
    dtSquared = DT * DT;

    // variable to measure execution time
    struct timeval time_start;
    struct timeval time_end;

    // get the start time
    gettimeofday(&time_start, NULL);

    // Copia os arrays prev_base, next_base e vel_base para a GPU
    hipMemcpy(dev_prev_base, prev_base, rows * cols * sizeof(float), cudaMencpyHostToDevice);
    hipMemcpy(dev_next_base, next_base, rows * cols * sizeof(float), cudaMencpyHostToDevice);
    hipMemcpy(dev_vel_base, vel_base, rows * cols * sizeof(float), cudaMencpyHostToDevice);

    // Chamada para função na GPU
    compute_wave<<<rows,1>>>(dev_prev_base,dev_next_base,dev_vel_base);

    // Sincronização de threads
    hipDeviceSynchronize();

    // Erro
    syncErr = hipGetLastError();
    asyncErr = hipDeviceSynchronize();
    if (syncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(syncErr));
    if (asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

    // Copia os arrays da GPU para a CPU
    hipMemcpy(prev_base, dev_prev_base, rows * cols * sizeof(float), cudaMencpyDeviceToHost);
    hipMemcpy(next_base, dev_next_base, rows * cols * sizeof(float), cudaMencpyDeviceToHost);
    hipMemcpy(vel_base, dev_vel_base, rows * cols * sizeof(float), cudaMencpyDeviceToHost);

    // get the end time
    gettimeofday(&time_end, NULL);

    double exec_time = (double) (time_end.tv_sec - time_start.tv_sec) + (double) (time_end.tv_usec - time_start.tv_usec) / 1000000.0;

    save_grid(rows, cols, next_base);

    printf("Iterations completed in %f seconds \n", exec_time);

    hipFree(dev_prev_base);
    hipFree(dev_next_base);
    hipFree(dev_vel_base);

    return 0;
}